/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update(void);
void printfinal(void);

int nsteps,                  /* number of time steps */
    tpoints,                 /* total points along string */
    rcode;                   /* generic return code */
float values[MAXPOINTS + 1], /* values at time t */
    oldval[MAXPOINTS + 1];   /* values at time (t-dt) */

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void) {
  char tchar[20];

  /* check number of points, number of iterations */
  while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
    printf("Enter number of points along vibrating string [%d-%d]: ", MINPOINTS,
           MAXPOINTS);
    scanf("%s", tchar);
    tpoints = atoi(tchar);
    if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
      printf("Invalid. Please enter value between %d and %d\n", MINPOINTS,
             MAXPOINTS);
  }
  while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
    printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
    scanf("%s", tchar);
    nsteps = atoi(tchar);
    if ((nsteps < 1) || (nsteps > MAXSTEPS))
      printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
  }

  printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void) {
  int i, j;
  float x, fac, k, tmp;

  /* Calculate initial values based on sine curve */
  fac = 2.0 * PI;
  k = 0.0;
  tmp = tpoints - 1;
  for (j = 1; j <= tpoints; j++) {
    x = k / tmp;
    values[j] = sin(fac * x);
    k = k + 1.0;
  }

  /* Initialize old values array */
  for (i = 1; i <= tpoints; i++) oldval[i] = values[i];
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void updateKernel(float *oldvald, float *valuesd, int nsteps, int tpoints) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float newval;
  for (int i = 1; i <= nsteps; ++i) {
    if (idx == 1 || idx == tpoints)
      newval = 0.0;
    else
      newval = 1.82 * valuesd[idx] - oldvald[idx];
    oldvald[idx] = valuesd[idx];
    valuesd[idx] = newval;
  }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal() {
  int i;

  for (i = 1; i <= tpoints; i++) {
    printf("%6.4f ", values[i]);
    if (i % 10 == 0) printf("\n");
  }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[]) {
  float *oldvald, *valuesd;
  sscanf(argv[1], "%d", &tpoints);
  sscanf(argv[2], "%d", &nsteps);
  check_param();
  printf("Initializing points on the line...\n");
  init_line();
  printf("Updating all points for all time steps...\n");

  int blocksize = 1024;
  int gridsize = (tpoints + blocksize) / blocksize;
  int size = blocksize * gridsize * sizeof(float);
  hipMalloc(&oldvald, size);
  hipMalloc(&valuesd, size);
  hipMemcpy(oldvald, oldval, size, hipMemcpyHostToDevice);
  hipMemcpy(valuesd, values, size, hipMemcpyHostToDevice);

  updateKernel<<<gridsize, blocksize>>>(oldvald, valuesd, nsteps, tpoints);
  hipMemcpy(values, valuesd, size, hipMemcpyDeviceToHost);

  hipFree(oldvald);
  hipFree(valuesd);

  printf("Printing final results...\n");
  printfinal();
  printf("\nDone.\n\n");

  return 0;
}
