#include <hip/hip_runtime.h>
#include <iostream>

#define WIDTH 15
#define TILE_WIDTH 5

void MatrixMulOnDevice(float *M, float *N, float *P, int Width);
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width);
void PrintMatrix(float *X, int Width, char ch);

int main() {
  float A[WIDTH * WIDTH];
  float B[WIDTH * WIDTH];
  float C[WIDTH * WIDTH];
  for (int i = 0; i < WIDTH; ++i) {
    for (int j = 0; j < WIDTH; ++j) {
      A[i * WIDTH + j] = i * WIDTH + j;
      B[i * WIDTH + j] = i * WIDTH + j;
    }
  }
  MatrixMulOnDevice(A, B, C, WIDTH);
  PrintMatrix(A, WIDTH, 'A');
  PrintMatrix(B, WIDTH, 'B');
  PrintMatrix(C, WIDTH, 'C');
}

void MatrixMulOnDevice(float *M, float *N, float *P, int Width) {
  int size = Width * Width * sizeof(float);
  float *Md, *Nd, *Pd;
  hipMalloc(&Md, size);
  hipMalloc(&Nd, size);
  hipMalloc(&Pd, size);

  hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
  hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

  dim3 dimGrid(Width / TILE_WIDTH, Width / TILE_WIDTH);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);

  hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);
}

__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float Pvalue = 0;
  for (int k = 0; k < Width; ++k) {
    Pvalue += Md[row * Width + k] * Nd[k * Width + col];
  }
  Pd[row * Width + col] = Pvalue;
}

void PrintMatrix(float *X, int Width, char ch) {
  std::cout << ch << ":" << std::endl;
  for (int i = 0; i < Width; ++i) {
    for (int j = 0; j < Width; ++j) {
      std::cout << X[i * Width + j] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}
