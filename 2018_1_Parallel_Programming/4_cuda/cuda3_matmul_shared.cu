#include <hip/hip_runtime.h>
#include <iostream>

#define WIDTH 15
#define TILE_WIDTH 5

void MatrixMulOnDevice(float *M, float *N, float *P, int Width);
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width);
void PrintMatrix(float *X, int Width, char ch);

int main() {
  float A[WIDTH * WIDTH];
  float B[WIDTH * WIDTH];
  float C[WIDTH * WIDTH];
  for (int i = 0; i < WIDTH; ++i) {
    for (int j = 0; j < WIDTH; ++j) {
      A[i * WIDTH + j] = i * WIDTH + j;
      B[i * WIDTH + j] = i * WIDTH + j;
    }
  }
  MatrixMulOnDevice(A, B, C, WIDTH);
  PrintMatrix(A, WIDTH, 'A');
  PrintMatrix(B, WIDTH, 'B');
  PrintMatrix(C, WIDTH, 'C');
}

void MatrixMulOnDevice(float *M, float *N, float *P, int Width) {
  int size = Width * Width * sizeof(float);
  float *Md, *Nd, *Pd;
  hipMalloc(&Md, size);
  hipMalloc(&Nd, size);
  hipMalloc(&Pd, size);

  hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
  hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

  dim3 dimGrid(Width / TILE_WIDTH, Width / TILE_WIDTH);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);

  hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);
}

__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width) {
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = blockIdx.y * blockDim.y + ty;
  int col = blockIdx.x * blockDim.x + tx;

  float Pvalue = 0;
  for (int m = 0; m < Width / TILE_WIDTH; ++m) {
    Mds[ty][tx] = Md[row * Width + (m * TILE_WIDTH + tx)];
    Nds[ty][tx] = Nd[col + (m * TILE_WIDTH + ty) * Width];
    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }
  Pd[row * Width + col] = Pvalue;
}

void PrintMatrix(float *X, int Width, char ch) {
  std::cout << ch << ":" << std::endl;
  for (int i = 0; i < Width; ++i) {
    for (int j = 0; j < Width; ++j) {
      std::cout << X[i * Width + j] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}
